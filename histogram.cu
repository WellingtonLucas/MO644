#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255
#define TILE_WIDTH 16

typedef struct {
	unsigned char red, green, blue;
} PPMPixel;

typedef struct {
	int x, y;
	PPMPixel *data;
} PPMImage;

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

//funcao que le o imagem ppm
static PPMImage *readPPM(const char *filename);

__global__ void device_histogram(PPMPixel *image ,float *h, int *l, int *c){
	
	int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
	
	int n = *l * *c;
	
	int j = threadIdx.x*blockDim.x + threadIdx.y;
	
	int x, y, z;
	
	__shared__ float h_private[64];
	
	//inicializando o contador do bin na copia privada de h
	if(j < 64){
		h_private[j] = 0;
	}
	__syncthreads();

	if(row < *l && col < *c){
		for (x = 0; x <= 3; x++) {
			for (y = 0; y <= 3; y++) {
				for (z = 0; z <= 3; z++) {
					if (image[*c*row + col].red == x && image[*c*row + col].green == y 
						&& image[*c*row + col].blue == z) {
						atomicAdd(&(h_private[x*16+y*4+z]), 1.0f);
					}
				}
			}
		}
		
	}

	__syncthreads();

	//adicionando em h com a normalizacao
	if(j < 64){
		atomicAdd(&(h[j]), h_private[j]/n);
	}
	
}

void Histogram(PPMImage *image, float *h) {
	
	int rows, cols, i;
	int *d_r, *d_c;
	float *d_h;
	PPMPixel *d_image;	

	float n = image->y * image->x;

	cols = image->x;
	rows = image->y;

	
	size_t bytes = sizeof(float)*64;

	for (i = 0; i < n; i++) {
		image->data[i].red = floor((image->data[i].red * 4) / 256);
		image->data[i].blue = floor((image->data[i].blue * 4) / 256);
		image->data[i].green = floor((image->data[i].green * 4) / 256);
	}


	hipMalloc((void**)&d_r, sizeof(int));
	hipMalloc((void**)&d_c, sizeof(int));
	hipMalloc((void**)&d_h, bytes);
	hipMalloc((void**)&d_image, sizeof(PPMPixel)*cols*rows);

	dim3 dimGrid(ceil((float) cols/TILE_WIDTH), ceil((float) rows/TILE_WIDTH ), 1);//numero de blocos de threads
   	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1); //numero de threads por bloco
    
	hipMemcpy(d_h, h, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_image, image->data, sizeof(PPMPixel)*cols*rows, hipMemcpyHostToDevice);
	hipMemcpy(d_r, &rows, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, &cols, sizeof(int), hipMemcpyHostToDevice);
	
	device_histogram<<<dimGrid , dimBlock>>>(d_image, d_h, d_r, d_c);
	
	hipMemcpy(h, d_h, bytes, hipMemcpyDeviceToHost);
   
    hipFree(d_c);
    hipFree(d_r); 
    hipFree(d_h);
    hipFree(d_image);
}


int main(int argc, char *argv[]) {

	if( argc != 2 ) {
		printf("Too many or no one arguments supplied.\n");
	}

	double t_start, t_end;
	int i;
	char *filename = argv[1]; //Recebendo o arquivo!;
	
	PPMImage *image = readPPM(filename);

	float *h = (float*)malloc(sizeof(float) * 64);

	//Inicializar h
	for(i=0; i < 64; i++) h[i] = 0.0;

	t_start = rtclock();
	Histogram(image, h);
	t_end = rtclock();

	for (i = 0; i < 64; i++){
		printf("%0.3f ", h[i]);
	}
	printf("\n");
	fprintf(stdout, "\n%0.6lfs\n", t_end - t_start);  
	free(h);

	return 0;
}

static PPMImage *readPPM(const char *filename) {
	char buff[16];
	PPMImage *img;
	FILE *fp;
	int c, rgb_comp_color;
	fp = fopen(filename, "rb");
	if (!fp) {
		fprintf(stderr, "Unable to open file '%s'\n", filename);
		exit(1);
	}

	if (!fgets(buff, sizeof(buff), fp)) {
		perror(filename);
		exit(1);
	}

	if (buff[0] != 'P' || buff[1] != '6') {
		fprintf(stderr, "Invalid image format (must be 'P6')\n");
		exit(1);
	}

	img = (PPMImage *) malloc(sizeof(PPMImage));
	if (!img) {
		fprintf(stderr, "Unable to allocate memory\n");
		exit(1);
	}

	c = getc(fp);
	while (c == '#') {
		while (getc(fp) != '\n')
			;
		c = getc(fp);
	}

	ungetc(c, fp);
	if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
		fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
		exit(1);
	}

	if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
		fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
				filename);
		exit(1);
	}

	if (rgb_comp_color != RGB_COMPONENT_COLOR) {
		fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
		exit(1);
	}

	while (fgetc(fp) != '\n');
	img->data = (PPMPixel*) malloc(img->x * img->y * sizeof(PPMPixel));

	if (!img) {
		fprintf(stderr, "Unable to allocate memory\n");
		exit(1);
	}

	if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
		fprintf(stderr, "Error loading image '%s'\n", filename);
		exit(1);
	}

	fclose(fp);
	return img;
}
/*
entrada, tempo_serial, tempo_GPU_criar_buffer, tempo_GPU_offload_enviar, tempo_kernel, tempo_GPU_offload_receber, GPU_total, speedup 
arq1.ppm, 0.218407, 0.112063, 0.008347, 0.000029, 0.003583, 0.121043, 1.804375305
arq2.ppm, 0.410912, 0.146193, 0.018426, 0.000035, 0.013859, 0.139981, 2.935484101 
arq3.ppm, 1.532259, 0.110749, 0.072426, 0.000042, 0.055907, 0.239100, 6.408444166


*/